// 文件：bfs_cuda.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <limits>

// CUDA kernel: 扫描当前 frontier，松弛所有邻居，构建 next frontier
__global__
void bfs_kernel(const int *row_ptr,
                const int *col_ind,
                const int  *frontier,
                int          frontier_size,
                int        *dist,
                int        *next_frontier,
                int        *next_frontier_size,
                int          current_dist)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= frontier_size) return;

    int u = frontier[idx];
    int start = row_ptr[u];
    int end   = row_ptr[u+1];
    for (int ei = start; ei < end; ++ei) {
        int v = col_ind[ei];
        // 原子判断并设置距离
        if (atomicCAS(&dist[v], -1, current_dist+1) == -1) {
            // 只有当 dist[v] 原先为 -1 时，才算新加入 frontier
            int pos = atomicAdd(next_frontier_size, 1);
            next_frontier[pos] = v;
        }
    }
}

void bfs_cuda(int n,
              const std::vector<int>& row_ptr_h,
              const std::vector<int>& col_ind_h,
              int source,
              std::vector<int>& dist_h)
{
    // 1) 申请并拷贝图到设备
    int *d_row_ptr, *d_col_ind;
    hipMalloc(&d_row_ptr, (n+1)*sizeof(int));
    hipMalloc(&d_col_ind, col_ind_h.size()*sizeof(int));
    hipMemcpy(d_row_ptr, row_ptr_h.data(), (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, col_ind_h.data(), col_ind_h.size()*sizeof(int), hipMemcpyHostToDevice);

    // 2) 申请并初始化 distance 数组
    int *d_dist;
    hipMalloc(&d_dist, n*sizeof(int));
    hipMemset(d_dist, -1, n*sizeof(int));

    // 3) 申请 frontier 和 next_frontier 空间
    int *d_frontier, *d_next_frontier;
    hipMalloc(&d_frontier, n*sizeof(int));
    hipMalloc(&d_next_frontier, n*sizeof(int));

    // 4) 申请并初始化 frontier 大小变量
    int *d_frontier_size, *d_next_frontier_size;
    hipMalloc(&d_frontier_size, sizeof(int));
    hipMalloc(&d_next_frontier_size, sizeof(int));

    // 5) 初始化第 0 轮 frontier
    int h_frontier_size = 1;
    hipMemcpy(d_frontier_size, &h_frontier_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier, &source, sizeof(int), hipMemcpyHostToDevice);
    int zero = 0;
    hipMemcpy(d_next_frontier_size, &zero, sizeof(int), hipMemcpyHostToDevice);
    // source 的距离置为 0
    hipMemcpy(&d_dist[source], &zero, sizeof(int), hipMemcpyHostToDevice);

    int current_dist = 0;
    // 6) 迭代，多轮 kernel launch
    while (true) {
        // 读出当前 frontier_size
        hipMemcpy(&h_frontier_size, d_frontier_size, sizeof(int), hipMemcpyDeviceToHost);
        if (h_frontier_size == 0) break;

        int threads = 256;
        int blocks  = (h_frontier_size + threads - 1) / threads;
        // 每轮开始前，清 next_frontier_size
        hipMemcpy(d_next_frontier_size, &zero, sizeof(int), hipMemcpyHostToDevice);

        // 运行 BFS kernel
        bfs_kernel<<<blocks, threads>>>(
            d_row_ptr, d_col_ind,
            d_frontier, h_frontier_size,
            d_dist,
            d_next_frontier, d_next_frontier_size,
            current_dist
        );
        hipDeviceSynchronize();

        // 交换 frontier 指针与大小
        std::swap(d_frontier, d_next_frontier);
        hipMemcpy(d_frontier_size, d_next_frontier_size, sizeof(int), hipMemcpyDeviceToHost);

        // 下一轮距离递增
        ++current_dist;
    }

    // 7) 将结果拷回 host
    dist_h.resize(n);
    hipMemcpy(dist_h.data(), d_dist, n*sizeof(int), hipMemcpyDeviceToHost);

    // 8) 释放资源
    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_dist);
    hipFree(d_frontier);
    hipFree(d_next_frontier);
    hipFree(d_frontier_size);
    hipFree(d_next_frontier_size);
}

int main()
{
    // 示例：构造一个简单无向图
    // 0—1—2
    // |  |
    // 3—4
    int n = 5;
    std::vector<int> row_ptr = {0, 2, 5, 7, 9, 10};
    std::vector<int> col_ind = {
        1,3,    // 0
        0,2,4,  // 1
        1,4,    // 2
        0,4,    // 3
        1,2,3   // 4
    };

    int source = 0;
    std::vector<int> dist;
    bfs_cuda(n, row_ptr, col_ind, source, dist);

    std::cout << "Distances from node " << source << ":\n";
    for (int i = 0; i < n; ++i)
        std::cout << "  to " << i << " = " << dist[i] << "\n";
    return 0;
}