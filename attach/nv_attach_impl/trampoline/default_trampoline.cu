#include <__clang_cuda_builtin_vars.h>
#include <atomic>
#include <chrono>
#include <csignal>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iterator>
#include <ostream>
#include <stdio.h>
#include <string>
#include <thread>
#include <vector>

/* clang++-17 -S ./default_trampoline.cu -Wall --cuda-gpu-arch=sm_60 -O2
 * -L/usr/local/cuda/lib64/ -lcudart*/
enum class HelperOperation {
	MAP_LOOKUP = 1,
	MAP_UPDATE = 2,
	MAP_DELETE = 3,
	MAP_GET_NEXT_KEY = 4,
	TRACE_PRINTK = 6,
	GET_CURRENT_PID_TGID = 14,
	PUTS = 501
};

union HelperCallRequest {
	struct {
		char key[1 << 30];
	} map_lookup;
	struct {
		char key[1 << 30];
		char value[1 << 30];
		uint64_t flags;
	} map_update;
	struct {
		char key[1 << 30];
	} map_delete;

	struct {
		char fmt[1000];
		int fmt_size;
		unsigned long arg1, arg2, arg3;
	} trace_printk;
	struct {
		char data[10000];
	} puts;
	struct {
	} get_tid_pgid;
};

union HelperCallResponse {
	struct {
		int result;
	} map_update, map_delete, trace_printk, puts;
	struct {
		const void *value;
	} map_lookup;
	struct {
		uint64_t result;
	} get_tid_pgid;
};
/**
 * 我们在这块结构体里放两个标志位和一个简单的参数字段
 * - flag1: device -> host 的信号，“我有请求要处理”
 * - flag2: host   -> device 的信号，“我处理完了”
 * - paramA: 设备端写入的参数，让主机端使用
 */
struct CommSharedMem {
	int flag1;
	int flag2;
	int occupy_flag;
	int request_id;
	long map_id;
	HelperCallRequest req;
	HelperCallResponse resp;
	uint64_t time_sum[8];
};

const int BPF_MAP_TYPE_NV_GPU_ARRAY_MAP = 1502;
const int BPF_MAP_TYPE_NV_GPU_RINGBUF_MAP = 1527;

struct MapBasicInfo {
	bool enabled;
	int key_size;
	int value_size;
	int max_entries;
	int map_type;
	void *extra_buffer;
	uint64_t max_thread_count;
};
__device__ __forceinline__ uint64_t read_globaltimer()
{
	uint64_t timestamp;
	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(timestamp));
	return timestamp;
}

__constant__ uintptr_t constData;
__constant__ MapBasicInfo map_info[256];
extern "C" __device__ void spin_lock(volatile int *lock)
{
	while (atomicCAS((int *)lock, 0, 1) == 1) {
		// 自旋等待锁变为可用
	}
	// printf("lock acquired by %d\n", threadIdx.x + blockIdx.x *
	// blockDim.x);
}

extern "C" __device__ void spin_unlock(int *lock)
{
	atomicExch(lock, 0); // 将锁标志重置为 0
	// printf("lock released by %d\n", threadIdx.x + blockIdx.x *
	// blockDim.x);
}
extern "C" __device__ HelperCallResponse make_helper_call(long map_id,
							  int req_id)
{
	CommSharedMem *g_data = (CommSharedMem *)constData;
	int lane_id = threadIdx.x & 31;
	HelperCallResponse my_resp = {};

	for (int active_lane = 0; active_lane < 32; active_lane++) {
		unsigned int active_mask = __activemask();
		bool lane_is_active = (active_mask >> active_lane) & 1;

		if (lane_is_active && lane_id == active_lane) {
			spin_lock(&g_data->occupy_flag);

			int val = 42;
			g_data->request_id = req_id;
			g_data->map_id = map_id;

			asm volatile(".reg .pred p0;                   \n\t"
				     "membar.sys;                      \n\t"
				     "st.global.u32 [%1], 1;           \n\t"
				     "spin_wait:                       \n\t"
				     "membar.sys;                      \n\t"
				     "ld.global.u32 %0, [%2];          \n\t"
				     "setp.eq.u32 p0, %0, 0;           \n\t"
				     "@p0 bra spin_wait;               \n\t"
				     "st.global.u32 [%2], 0;           \n\t"
				     "membar.sys;                      \n\t"
				     :
				     : "r"(val), "l"(&g_data->flag1),
				       "l"(&g_data->flag2)
				     : "memory");

			my_resp = g_data->resp;

			spin_unlock(&g_data->occupy_flag);
		}

		__syncwarp(active_mask);
	}

	return my_resp;
}
extern "C" __device__ inline void simple_memcpy(void *dst, void *src, int sz)
{
	for (int i = 0; i < sz; i++)
		((char *)dst)[i] = ((char *)src)[i];
}

__device__ uint64_t getGlobalThreadId()
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	int width = gridDim.x * blockDim.x;
	int height = gridDim.y * blockDim.y;
	return ((uint64_t)z * width * height) + (y * width) + x;
}
__device__ void *array_map_offset(uint64_t idx, const MapBasicInfo &info)
{
	return (void *)((uintptr_t)info.extra_buffer +
			idx * info.max_thread_count * info.value_size +
			getGlobalThreadId() * info.value_size);
}

extern "C" __noinline__ __device__ uint64_t _bpf_helper_ext_0001(
	uint64_t map, uint64_t key, uint64_t a, uint64_t b, uint64_t c)
{
	CommSharedMem *global_data = (CommSharedMem *)constData;
	auto &req = global_data->req;
	// CallRequest req;
	const auto &map_info = ::map_info[map];
	if (map_info.map_type == BPF_MAP_TYPE_NV_GPU_ARRAY_MAP) {
		auto real_key = *(uint32_t *)(uintptr_t)key;
		auto offset = array_map_offset(real_key, map_info);
		return (uint64_t)offset;
	}
	// printf("helper1 map %ld keysize=%d valuesize=%d\n", map,
	//        map_info.key_size, map_info.value_size);
	simple_memcpy(&req.map_lookup.key, (void *)(uintptr_t)key,
		      map_info.key_size);

	HelperCallResponse resp =
		make_helper_call((long)map, (int)HelperOperation::MAP_LOOKUP);

	return (uintptr_t)resp.map_lookup.value;
}

extern "C" __noinline__ __device__ uint64_t _bpf_helper_ext_0002(
	uint64_t map, uint64_t key, uint64_t value, uint64_t flags, uint64_t a)
{
	CommSharedMem *global_data = (CommSharedMem *)constData;
	auto &req = global_data->req;
	const auto &map_info = ::map_info[map];
	if (map_info.map_type == BPF_MAP_TYPE_NV_GPU_ARRAY_MAP) {
		auto real_key = *(uint32_t *)(uintptr_t)key;
		auto offset = array_map_offset(real_key, map_info);
		simple_memcpy(offset, (void *)(uintptr_t)value,
			      map_info.value_size);
		return 0;
	}
	// printf("helper2 map %ld keysize=%d
	// valuesize=%d\n",map,map_info.key_size,map_info.value_size);
	simple_memcpy(&req.map_update.key, (void *)(uintptr_t)key,
		      map_info.key_size);
	simple_memcpy(&req.map_update.value, (void *)(uintptr_t)value,
		      map_info.value_size);
	req.map_update.flags = (uintptr_t)flags;

	HelperCallResponse resp =
		make_helper_call((long)map, (int)HelperOperation::MAP_UPDATE);
	return resp.map_update.result;
}

extern "C" __noinline__ __device__ uint64_t _bpf_helper_ext_0003(
	uint64_t map, uint64_t key, uint64_t a, uint64_t b, uint64_t c)
{
	CommSharedMem *global_data = (CommSharedMem *)constData;
	auto &req = global_data->req;
	const auto &map_info = ::map_info[map];
	// printf("helper3 map %ld keysize=%d
	// valuesize=%d\n",map,map_info.key_size,map_info.value_size);
	simple_memcpy(&req.map_delete.key, (void *)(uintptr_t)key,
		      map_info.key_size);
	HelperCallResponse resp =
		make_helper_call((long)map, (int)HelperOperation::MAP_DELETE);
	return resp.map_delete.result;
}

extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0006(uint64_t fmt, uint64_t fmt_size, uint64_t arg1,
		     uint64_t arg2, uint64_t arg3)
{
	// printf("Calling 0006 fmt %s\n",(char*)fmt);
	CommSharedMem *global_data = (CommSharedMem *)constData;
	auto &req = global_data->req;
	char *out = (char *)req.trace_printk.fmt;
	char *in = (char *)(uintptr_t)fmt;
	for (auto i = 0; i < fmt_size; i++) {
		out[i] = in[i];
	}
	req.trace_printk.fmt_size = fmt_size;
	req.trace_printk.arg1 = arg1;
	req.trace_printk.arg2 = arg2;
	req.trace_printk.arg3 = arg3;
	HelperCallResponse resp =
		make_helper_call(0, (int)HelperOperation::TRACE_PRINTK);
	return resp.trace_printk.result;
}

extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0014(uint64_t, uint64_t, uint64_t, uint64_t, uint64_t)
{
	HelperCallResponse resp =
		make_helper_call(0, (int)HelperOperation::GET_CURRENT_PID_TGID);
	return resp.get_tid_pgid.result;
}
struct ringbuf_header {
	uint64_t head;
	uint64_t tail;
	int dirty;
};

// perf event output
extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0025(uint64_t ctx, uint64_t map, uint64_t flags, uint64_t data,
		     uint64_t data_size)
{
	const auto &map_info = ::map_info[map];
	if (map_info.map_type == BPF_MAP_TYPE_NV_GPU_RINGBUF_MAP) {
		// printf("Starting perf output, value size=%d, max entries = %d\n",
		//        map_info.value_size, map_info.max_entries);
		auto entry_size = sizeof(ringbuf_header) +
				  map_info.max_entries * (sizeof(uint64_t) +
							  map_info.value_size);
		auto header =
			(ringbuf_header *)(uintptr_t)(getGlobalThreadId() *
							      entry_size +
						      (char *)map_info
							      .extra_buffer);
		// printf("header->head=%lu, header->tail=%lu\n", header->head,
		//        header->tail);
		if (header->tail - header->head == map_info.max_entries) {
			// Buffer is full
			// printf("Buffer is full\n");
			return 2;
		}
		header->dirty = 1;
		auto tail_to_put =
			__atomic_fetch_add(&header->tail, 1, __ATOMIC_SEQ_CST);
		auto real_tail = tail_to_put % map_info.max_entries;
		// printf("real tail=%lu\n", real_tail);
		auto buffer =
			((char *)header) + sizeof(ringbuf_header) +
			real_tail * (sizeof(uint64_t) + map_info.value_size);
		// printf("before wrtting size to %lx, of %lu\n",
		//        (uintptr_t)buffer, data_size);
		*(uint64_t *)(uintptr_t)buffer = data_size;
		// printf("before copying..\n");
		simple_memcpy(buffer + sizeof(uint64_t),
			      (void *)(uintptr_t)data, data_size);
		// printf("data copied\n");
		header->dirty = 0;
		// printf("Generated %d bytes of data\n", (int)data_size);
		return 0;

	} else {
		printf("Calling bpf_perf_event_output on unsupported map!");
		return 1;
	}
}

extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0501(uint64_t data, uint64_t, uint64_t, uint64_t, uint64_t)
{
	CommSharedMem *global_data = (CommSharedMem *)constData;
	auto &req = global_data->req.puts;

	const char *input = (const char *)data;
	int idx = 0;
	while (input[idx]) {
		req.data[idx] = input[idx];
		idx++;
	}
	req.data[idx] = 0;
	HelperCallResponse resp =
		make_helper_call(0, (int)HelperOperation::PUTS);
	return resp.puts.result;
}

extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0502(uint64_t, uint64_t, uint64_t, uint64_t, uint64_t)
{
	return read_globaltimer();
}

extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0503(uint64_t x, uint64_t y, uint64_t z, uint64_t, uint64_t)
{
	// get block idx
	*(uint64_t *)(uintptr_t)x = blockIdx.x;
	*(uint64_t *)(uintptr_t)y = blockIdx.y;
	*(uint64_t *)(uintptr_t)z = blockIdx.z;

	return 0;
}
extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0504(uint64_t x, uint64_t y, uint64_t z, uint64_t, uint64_t)
{
	// get block dim
	*(uint64_t *)(uintptr_t)x = blockDim.x;
	*(uint64_t *)(uintptr_t)y = blockDim.y;
	*(uint64_t *)(uintptr_t)z = blockDim.z;

	return 0;
}
extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0505(uint64_t x, uint64_t y, uint64_t z, uint64_t, uint64_t)
{
	// get threadIdx
	*(uint64_t *)(uintptr_t)x = threadIdx.x;
	*(uint64_t *)(uintptr_t)y = threadIdx.y;
	*(uint64_t *)(uintptr_t)z = threadIdx.z;

	return 0;
}
extern "C" __noinline__ __device__ uint64_t
_bpf_helper_ext_0506(uint64_t, uint64_t, uint64_t, uint64_t, uint64_t)
{
	asm("membar.sys;                      \n\t");
	return 0;
}

extern "C" __global__ void bpf_main(void *mem, size_t sz)
{
	printf("kernel function entered, mem=%lx, memsz=%ld\n", (uintptr_t)mem,
	       sz);
	char buf[16] = "aaa";
	printf("setup function, const data=%lx\n", constData);
	auto result = _bpf_helper_ext_0001(1ull << 32, (uintptr_t)buf, 0, 0, 0);
	_bpf_helper_ext_0002(1ull << 32, (uintptr_t)buf, (uintptr_t)buf, 0, 0);
	_bpf_helper_ext_0003(1ull << 32, (uintptr_t)buf, 0, 0, 0);
	const char msg[] = "Message from bpf: %d, %lx";
	_bpf_helper_ext_0006((uint64_t)(uintptr_t)msg, sizeof(msg), 10, 20, 0);

	printf("call done\n");
	printf("got response %d at %d\n", *(int *)result,
	       threadIdx.x + blockIdx.x * blockDim.x);
	*(int *)mem = 123;
}

static std::atomic<bool> should_exit;
void signal_handler(int)
{
	should_exit.store(true);
}
int main()
{
	signal(SIGINT, signal_handler);

	// 1. 先在主机上分配一块普通内存
	CommSharedMem *hostMem = (CommSharedMem *)malloc(sizeof(CommSharedMem));
	if (!hostMem) {
		std::cerr << "Failed to allocate hostMem\n";
		return -1;
	}

	// 2. 注册成 pinned memory (可被GPU直接访问)
	hipError_t err = hipHostRegister(hostMem, sizeof(CommSharedMem),
					   hipHostRegisterMapped);
	if (err != hipSuccess) {
		std::cerr
			<< "hipHostRegister error: " << hipGetErrorString(err)
			<< "\n";
		free(hostMem);
		return -1;
	}

	// 3. 获取对应的设备指针(这样DeviceKernel就能直接访问这个地址)
	CommSharedMem *devPtr = nullptr;
	err = hipHostGetDevicePointer((void **)&devPtr, (void *)hostMem, 0);
	if (err != hipSuccess) {
		std::cerr << "hipHostGetDevicePointer error: "
			  << hipGetErrorString(err) << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	printf("dev ptr should be %lx, host ptr is %lx\n", (uintptr_t)devPtr,
	       (uintptr_t)hostMem);
	err = hipMemcpyToSymbol(HIP_SYMBOL(constData), &devPtr, sizeof(CommSharedMem *));
	if (err != hipSuccess) {
		std::cerr << "hipMemcpyToSymbol error: "
			  << hipGetErrorString(err) << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	int buf = 11223344;
	err = hipHostRegister((void *)&buf, sizeof(buf),
			       hipHostRegisterMapped);
	if (err != hipSuccess) {
		std::cerr << "cudaHostRegister(2) error: "
			  << hipGetErrorString(err) << " " << err << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	char *devPtrStr = nullptr;
	err = hipHostGetDevicePointer((void **)&devPtrStr, (void *)&buf, 0);
	if (err != hipSuccess) {
		std::cerr << "cudaHostGetDevicePointer(2) error: "
			  << hipGetErrorString(err) << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	// 初始化标志位
	memset(hostMem, 0, sizeof(*hostMem));
	// 4. 启动一个线程, 模拟host侧的处理逻辑
	std::thread hostThread([&]() {
		std::cout << "[Host Thread] Start waiting...\n";

		// 这里简单用轮询，检测到flag1=1就处理
		while (!should_exit.load()) {
			if (hostMem->flag1 == 1) {
				// 清掉flag1防止重复处理
				hostMem->flag1 = 0;
				// 假设处理数据 paramA
				std::cout
					<< "[Host Thread] Got request: req_id="
					<< hostMem->request_id
					<< ", handling...\n";
				if (hostMem->request_id == 1) {
					std::cout << "call map_lookup="
						  << hostMem->req.map_lookup.key
						  << std::endl;
					// strcpy(hostMem->resp.map_lookup.value,
					//        "your value");
					hostMem->resp.map_lookup.value =
						devPtrStr;
				}
				// std::atomic_thread_fence(std::memory_order_seq_cst);

				// 处理完后, 把 flag2=1, 让设备端退出自旋
				hostMem->flag2 = 1;

				// 在实际开发中，可以加个内存栅栏，比如：
				std::atomic_thread_fence(
					std::memory_order_seq_cst);

				// 处理一次就退出本线程循环
				// break;
				std::cout << "handle done, timesum = "
					  << hostMem->time_sum[1] << std::endl;
			}

			// 为了演示，这里短暂休眠，避免100%占用CPU
			std::this_thread::sleep_for(
				std::chrono::milliseconds(10));
		}

		std::cout << "[Host Thread] Done.\n";
	});
	std::vector<MapBasicInfo> local_map_info(256);

	local_map_info[1].enabled = true;
	local_map_info[1].key_size = 16;
	local_map_info[1].value_size = 16;
	hipMemcpyToSymbol(HIP_SYMBOL(map_info), local_map_info.data(),
			   sizeof(MapBasicInfo) * local_map_info.size());
	// 5. 启动核函数 (只发1个block,1个thread做演示)
	bpf_main<<<1, 1>>>(hostMem, sizeof(*hostMem));

	// 等待核函数执行完毕
	hipDeviceSynchronize();

	// 等待host线程结束
	hostThread.join();

	// 6. 收尾：解绑 pinned memory 并释放
	hipHostUnregister(hostMem);
	free(hostMem);

	std::cout << "All done.\n";
	return 0;
}
